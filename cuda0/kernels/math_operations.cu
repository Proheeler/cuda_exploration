#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_fd_funcs.h"
#include <iostream>
#include <ctime>

template <typename T>
__global__ void VecAdd(T* A, T* B, T* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

template <typename T>
__global__ void MatAdd(T** A, T **B,
                       T** C)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}


void doVectorAddition(int* a ,int* b,int* c,int N)
{
    std::clock_t start;
    double duration;
    start = std::clock();

    // allocate memory on GPU
    CudaArray<int> dev_a(N);
    CudaArray<int> dev_b(N);
    CudaArray<int> dev_c(N);

    // copy 2 arrays to device memory
    dev_a.set(a,N);
    dev_b.set(b,N);

    // <<< first element is the # of parallel blocks to launch
    // second >>> the # of threads per block
    VecAdd<<<1, N>>>(dev_a.getData(), dev_b.getData(), dev_c.getData());

    // copy from device to host
    dev_c.get(c,N);
    //    hipMemcpy(c, dev_c.getData(), N * sizeof(N), hipMemcpyDeviceToHost);


    std::cout << "DONE INT" << "\n";
    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << "\n";
    }
    std::cout<<"printf: "<< duration <<'\n';

}

void doVectorAddition(float* a ,float* b,float* c,int N)
{
    std::clock_t start;
    double duration;
    start = std::clock();

    // allocate memory on GPU
    CudaArray<float> dev_a(N);
    CudaArray<float> dev_b(N);
    CudaArray<float> dev_c(N);

    // copy 2 arrays to device memory
    dev_a.set(a,N);
    dev_b.set(b,N);

    // <<< first element is the # of parallel blocks to launch
    // second >>> the # of threads per block
    VecAdd<<<1, N>>>(dev_a.getData(), dev_b.getData(), dev_c.getData());

    // copy from device to host
    dev_c.get(c,N);

    std::cout << "DONE FLOAT" << "\n";
    duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
    for (int i = 0; i < N; i++) {
        std::cout << a[i] << " + " << b[i] << " = " << c[i] << "\n";
    }
    std::cout<<"printf: "<< duration <<'\n';

}
