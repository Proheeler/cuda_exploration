#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_fd_funcs.h"
#include <iostream>
#include <ctime>

template <typename T>
__global__ void VecAdd(T* A, T* B, T* C)
{
    int i = threadIdx.x;
    C[i] = A[i] + B[i];
}

template <typename T>
__global__ void MatAdd(T** A, T **B,
                       T** C)
{
    int i = threadIdx.x;
    int j = threadIdx.y;
    C[i][j] = A[i][j] + B[i][j];
}


void doVectorAddition(int* a ,int* b,int* c,int N)
{
      std::clock_t start;
      double duration;
      start = std::clock();

//      int *dev_a;
//      int *dev_b;
//      int *dev_c;

      // allocate memory on GPU
      CudaSmartPtr<int> dev_a(N);
      CudaSmartPtr<int> dev_b(N);
      CudaSmartPtr<int> dev_c(N);
//      hipMalloc((void**)&dev_a, N * sizeof(int));
//      hipMalloc((void**)&dev_b, N * sizeof(int));
//      hipMalloc((void**)&dev_c, N * sizeof(int));

      // copy 2 arrays to device memory
      hipMemcpy(dev_a.get(), a, N * sizeof(N), hipMemcpyHostToDevice);
      hipMemcpy(dev_b.get(), b, N * sizeof(N), hipMemcpyHostToDevice);

      // <<< first element is the # of parallel blocks to launch
      // second >>> the # of threads per block
      VecAdd<<<1, N>>>(dev_a.get(), dev_b.get(), dev_c.get());

      // copy from device to host
      hipMemcpy(c, dev_c.get(), N * sizeof(N), hipMemcpyDeviceToHost);

//       for (int i = 0; i < N; i++) {
//           std::cout << a[i] << " + " << b[i] << " = " << c[i] << "\n";
//       }

      std::cout << "DONE INT" << "\n";
      duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
      std::cout<<"printf: "<< duration <<'\n';

//      hipFree(dev_a);
//      hipFree(dev_b);
//      hipFree(dev_c);
}

void doVectorAddition(float* a ,float* b,float* c,int N)
{
      std::clock_t start;
      double duration;
      start = std::clock();

      float *dev_a;
      float *dev_b;
      float *dev_c;

      // allocate memory on GPU
      hipMalloc((void**)&dev_a, N * sizeof(float));
      hipMalloc((void**)&dev_b, N * sizeof(float));
      hipMalloc((void**)&dev_c, N * sizeof(float));

      // copy 2 arrays to device memory
      hipMemcpy(dev_a, a, N * sizeof(N), hipMemcpyHostToDevice);
      hipMemcpy(dev_b, b, N * sizeof(N), hipMemcpyHostToDevice);

      // <<< first element is the # of parallel blocks to launch
      // second >>> the # of threads per block
      VecAdd<<<1, N>>>(dev_a, dev_b, dev_c);

      // copy from device to host
      hipMemcpy(c, dev_c, N * sizeof(N), hipMemcpyDeviceToHost);

//       for (int i = 0; i < N; i++) {
//           std::cout << a[i] << " + " << b[i] << " = " << c[i] << "\n";
//       }

      std::cout << "DONE FLOAT" << "\n";
      duration = ( std::clock() - start ) / (double) CLOCKS_PER_SEC;
      std::cout<<"printf: "<< duration <<'\n';

      hipFree(dev_a);
      hipFree(dev_b);
      hipFree(dev_c);
}
