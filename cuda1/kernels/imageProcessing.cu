#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_fd_funcs.h"
#include "../include/CudaArray.h"
#include <iostream>
#include <ctime>


__global__ void convertRGBtoGrayScale(uint8_t* src, uint8_t* dst,int width,int height, int channels)
{
    int x = threadIdx.x+ blockIdx.x* blockDim.x;
    int y = threadIdx.y+ blockIdx.y* blockDim.y;
    if(x < width && y < height) {
        int grayOffset= y*width + x;// one can think of the RGB image having
        int rgbOffset= grayOffset*channels;// CHANNEL times columns than the gray scale
        unsigned char r =  src[rgbOffset]; // red value for pixel
        unsigned char g = src[rgbOffset+ 2]; // green value for pixel
        unsigned char b = src[rgbOffset+ 3]; // blue value for pixel// perform the rescaling and store it// We multiply by floating point constants
        dst[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}

__global__ void blurKernel(uint8_t *out, uint8_t *in, int width, int height,int BLUR_SIZE=5) {
  int col = blockIdx.x * blockDim.x + threadIdx.x;
  int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height) {
    int pixVal = 0;
    int pixels = 0;

    // Get the average of the surrounding BLUR_SIZE x BLUR_SIZE box
    for (int blurrow = -BLUR_SIZE; blurrow < BLUR_SIZE + 1; ++blurrow) {
      for (int blurcol = -BLUR_SIZE; blurcol < BLUR_SIZE + 1; ++blurcol) {

        int currow = row + blurrow;
        int curcol = col + blurcol;
        // Verify we have a valid image pixel
        if (currow > -1 && currow < height && curcol > -1 &&
            curcol < width) {
          pixVal += in[currow * width + curcol];
          pixels++; // Keep track of number of pixels in the avg
        }
      }
    }

    // Write our new pixel value out
    out[row * width + col] = (unsigned char)(pixVal / pixels);
  }
}
void h_blurImage(uint8_t* src,uint8_t* dst,int TileSize,int && width, int&& height,int blurSize)
{
    assert(src);
    assert(dst);
    CudaArray<uint8_t> dev_src(width*height);
    CudaArray<uint8_t> dev_dst(width*height);
    dev_src.set(src,width*height);
    dev_dst.set(dst,width*height);
    const dim3 blockSize(TileSize, TileSize);
    const dim3 gridSize((width / TileSize) + 1, (height / TileSize) + 1);
    blurKernel<<<gridSize, blockSize>>>(dev_dst.getData(),dev_src.getData(), width,height,blurSize);
    dev_dst.get(dst,width*height);
}
void h_convertRGBtoGrayScale(uint8_t* src,uint8_t* dst,int TileSize,int && width, int&& height,int && channels){
    assert(src);
    assert(dst);
    CudaArray<uint8_t> dev_src(width*height*channels);
    CudaArray<uint8_t> dev_dst(width*height);
    dev_src.set(src,width*height*channels);
    dev_dst.set(dst,width*height);
    const dim3 blockSize(TileSize, TileSize);
    const dim3 gridSize((width / TileSize) + 1, (height / TileSize) + 1);
    convertRGBtoGrayScale<<<gridSize, blockSize>>>(dev_src.getData(), dev_dst.getData(), width,height,channels);
    dev_dst.get(dst,width*height);


}



