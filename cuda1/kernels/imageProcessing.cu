#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_fd_funcs.h"
#include "../include/CudaArray.h"
#include <iostream>
#include <ctime>


__global__ void convertRGBtoGrayScale(uint8_t* src, uint8_t* dst,int width,int height)
{
    int x = threadIdx.x+ blockIdx.x* blockDim.x;
    int y = threadIdx.y+ blockIdx.y* blockDim.y;
    if(x < width && y < height) {
        int grayOffset= y*width + x;// one can think of the RGB image having
        int rgbOffset= grayOffset*3;// CHANNEL times columns than the gray scale
        unsigned char r =  src[rgbOffset]; // red value for pixel
        unsigned char g = src[rgbOffset+ 2]; // green value for pixel
        unsigned char b = src[rgbOffset+ 3]; // blue value for pixel// perform the rescaling and store it// We multiply by floating point constants
        dst[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}
void h_convertRGBtoGrayScale(uint8_t* src,uint8_t* dst,int && width, int&& height){
    assert(src);
    assert(dst);
    CudaArray<uint8_t> dev_src(width*height*3);
    CudaArray<uint8_t> dev_dst(width*height);
    dev_src.set(src,width*height*3);
    dev_dst.set(dst,width*height);
    const int BS = 32;
    const dim3 blockSize(BS, BS);
    const dim3 gridSize((width / BS) + 1, (height / BS) + 1);
    convertRGBtoGrayScale<<<gridSize, blockSize>>>(dev_src.getData(), dev_dst.getData(), width,height);
    dev_dst.get(dst,width*height);


}



