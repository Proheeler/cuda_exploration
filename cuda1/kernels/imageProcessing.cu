#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "../include/cuda_fd_funcs.h"
#include "../include/CudaArray.h"
#include <iostream>
#include <ctime>


__global__ void convertRGBtoGrayScale(uint8_t* src, uint8_t* dst,int width,int height, int channels)
{
    int x = threadIdx.x+ blockIdx.x* blockDim.x;
    int y = threadIdx.y+ blockIdx.y* blockDim.y;
    if(x < width && y < height) {
        int grayOffset= y*width + x;// one can think of the RGB image having
        int rgbOffset= grayOffset*channels;// CHANNEL times columns than the gray scale
        unsigned char r =  src[rgbOffset]; // red value for pixel
        unsigned char g = src[rgbOffset+ 2]; // green value for pixel
        unsigned char b = src[rgbOffset+ 3]; // blue value for pixel// perform the rescaling and store it// We multiply by floating point constants
        dst[grayOffset] = 0.21f*r + 0.71f*g + 0.07f*b;
    }
}
void h_convertRGBtoGrayScale(uint8_t* src,uint8_t* dst,int TileSize,int && width, int&& height,int && channels){
    assert(src);
    assert(dst);
    CudaArray<uint8_t> dev_src(width*height*3);
    CudaArray<uint8_t> dev_dst(width*height);
    dev_src.set(src,width*height*3);
    dev_dst.set(dst,width*height);
//    const int BS = 32;
    const dim3 blockSize(TileSize, TileSize);
    const dim3 gridSize((width / TileSize) + 1, (height / TileSize) + 1);
    convertRGBtoGrayScale<<<gridSize, blockSize>>>(dev_src.getData(), dev_dst.getData(), width,height,channels);
    dev_dst.get(dst,width*height);


}



